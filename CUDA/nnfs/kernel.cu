#include "hip/hip_runtime.h"
﻿//#include ""

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* a, float* b, float* c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{

	VecAdd <<<1, N >>> (A, B, C);
}



